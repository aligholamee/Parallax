#include "hip/hip_runtime.h"

// Let it be.
#define _CRT_SECURE_NO_WARNINGS
#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

// CUDA Runtime
#include <hip/hip_runtime.h>
#include <>

void fill_array(int *a, size_t n);
void prefix_sum_serial(int *a, size_t n);
void print_array(int *a, size_t n);
float compute_mse(int *a, int *b, int n);
int compute_prefix_sum(int *a, size_t n);


// CUDA Kernel
__global__ void
prefixSumCUDA(int *a, size_t n)
{
	/* Implementation of Naive Hillis and Steele Algorithm*/

	int tId = blockIdx.x * blockDim.x + threadIdx.x;
	
	int end = ceil(log2((float)n));

	for (int offset = 0; offset < end; offset++) {

		if (tId >= n) continue;

		if (tId >= (1 << offset)) {
			a[tId] += a[tId - (1 << offset)];
		}
	}
}

__global__ void
prefixSumMap(int *a, size_t n)
{
	int addsPerThread = 5;
	int tId = blockIdx.x * blockDim.x + threadIdx.x * addsPerThread;
	
	// Each thread computes addition of 10 elements per block
	for (int i = 1; i < addsPerThread; i++) {

		if (tId + i >= n) break;			// Safety check

		a[tId + i] += a[tId + i - 1];
	}
}

__global__ void
prefixSumReduce(int *a, size_t n)
{
	int addsPerThread = 5;
	int tId = blockIdx.x * blockDim.x + threadIdx.x;

	if (tId >= n) return;			// Safety check

	// One thread in each block computes the final result
	if (tId % BLOCK_SIZE == 0 ) {	// First thread of each block
		for (int i = 1; i < BLOCK_SIZE; i++) {

			if (i >= n) break;		// Safety check

			int mergeStart = i * addsPerThread;
			int mergeEnd = mergeStart + addsPerThread;

			for (int j = mergeStart; j < mergeEnd; j++) {
				a[j] += a[mergeStart - 1];
			}
		}


	}

}


void scanCPU(float *f_out, float *f_in, int i_n)
{
	f_out[0] = 0;
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i - 1] + f_in[i - 1];
}

int main(int argc, char *argv[]) {
	// Input N
	size_t n = 0;
	printf("[-] Please enter N: ");
	scanf("%uld\n", &n);
	// Allocate memory for array
	int * a = (int *)malloc(n * sizeof a);
	// Fill array with numbers 1..n
	fill_array(a, n);
	// Print array
	// print_array(a, n);
	// Compute prefix sum
	// prefix_sum(a, n);

	// Create a copy of a for comparison
	int * b = (int *)malloc(n * sizeof b);
	for (int i = 0; i < n; i++) {
		b[i] = a[i];
	}

	// print_array(b, n);

	// Compute the serial prefix sum on b
	prefix_sum_serial(b, n);

	// Compute the parallel prefix sum on a
	compute_prefix_sum(a, n);

	// Find the computation error 
	float error = compute_mse(a, b, n);

	printf("Computation error is %lf\n", error);

	// Print array
	// print_array(a, n);

	// Free allocated memory
	free(a);

	system("pause");
	return EXIT_SUCCESS;
}


int compute_prefix_sum(int *a, size_t n) {

	// To be allocated on device memory
	int* d_A;
	int* d_result;

	// Handles errors
	hipError_t error;

	if (a == NULL)
	{
		fprintf(stderr, "Failed to allocate host vector a!\n");
		exit(EXIT_FAILURE);
	}

	// Allocate memory on device
	int memSize = n * sizeof(int);

	error = hipMalloc((void **)&d_A, memSize);

	if (error != hipSuccess) {
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Copy data from host to device
	error = hipMemcpy(d_A, a, n * sizeof(int), hipMemcpyHostToDevice);

	if (error != hipSuccess) {
		printf("hipMemcpy d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Execution parameters
	// dim3 gridSize(2, 1, 1);
	// dim3 blockSize = (1024, 1, 1);

	printf("Processing data on GPU...\n");

	// Setup CUDA events for timings
	hipEvent_t start;

	// Create the CUDA events
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Kernel launch 
	// dim3 gridDimensions(ceil((float)n / BLOCK_SIZE), 1, 1);
	// dim3 blockDimensions(BLOCK_SIZE, 1, 1);
	
	dim3 gridDimensions(ceil((float)n/BLOCK_SIZE), 1, 1);
	dim3 blockDimensions(BLOCK_SIZE, 1, 1);

	// prefixSumCUDA <<< gridDimensions, blockDimensions >> > (d_result, d_A, n);

	prefixSumMap << <gridDimensions, blockDimensions >> > (d_A, n);
	prefixSumReduce << < gridDimensions, blockDimensions >> > (d_A, n);

	// Check kernel launch
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}


	// Wait for the stop event to be completed by all threads
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Compute the elapsed time
	float elapsed_time = 0.0f;
	error = hipEventElapsedTime(&elapsed_time, start, stop);

	printf("Elapsed time in msec = %lf\n", elapsed_time);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	error = hipMemcpy(a, d_A, n * sizeof(int), hipMemcpyDeviceToHost);


	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_A);

	return EXIT_SUCCESS;
}
void prefix_sum_serial(int *a, size_t n) {
	int i;
	for (i = 1; i < n; ++i) {
		a[i] = a[i] + a[i - 1];
	}
}

void print_array(int *a, size_t n) {
	int i;
	printf("[-] array: ");
	for (i = 0; i < n; ++i) {
		printf("%d, ", a[i]);
	}
	printf("\b\b \n");
}

void fill_array(int *a, size_t n) {
	int i;
	for (i = 0; i < n; ++i) {
		a[i] = i + 1;
	}
}

float compute_mse(int *a, int *b, int n) {
	float err;

	for (int i = 0; i < n; i++) {
		err += pow(a[i] - b[i], 2);
	}

	return err;
}